#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

// Implementación de la suma y multiplicación de matrices, dim3, punteros dobles

#define N 6
#define THREADS_PER_BLOCK 16


__global__
void matrix_add(int** dd_mat_a,int** dd_mat_b,int** dd_mat_c, int n)
{

        int cols = threadIdx.x + blockIdx.x*blockDim.x;
        int fils = threadIdx.y + blockIdx.y*blockDim.y;

        if( fils <n && cols < n ){
        dd_mat_c[fils][cols] = dd_mat_a[fils][cols] + dd_mat_b[fils][cols];
        }

}

__global__
void matrix_mult(int** dd_mat_a,int** dd_mat_b,int** dd_mat_c, int n)
{
        int cols = threadIdx.x + blockIdx.x*blockDim.x;
        int fils = threadIdx.y + blockIdx.y*blockDim.y;

        int i;
        if(fils < n && cols < n)
        {
        dd_mat_c[fils][cols] = 0;
        for(i=0;i<n;i++)
        {
        dd_mat_c[fils][cols] += dd_mat_a[fils][i]*dd_mat_b[i][cols];
        }
        }
}


void create_host_matrix(int*** mat,int n, int m){
        *mat = (int** )malloc(sizeof(int*)*n);
        (*mat)[0] = (int* )malloc(sizeof(int)*n*m);
        int i;
        for(i=1;i<n;i++){
                (*mat)[i] = (*mat)[0]+i*m;
        }
}



void fill_host_matrix(int** mat, int n, int m){
        int i,j;
        for(i=0; i<n ;i++){
                for(j=0; j<m ;j++)
                        //mat[i][j] = rand()%2+1;
                        mat[i][j] = 1;
        }
}

void fill_zero(int** mat,int n, int m, int value=0){
        int i,j;
        for(i=0;i<n;i++)
                for(j=0;j<m;j++)
                        mat[i][j] = value;
}


void print(int** mat,int n, int m){
        int i,j;
        for(i=0; i<n ;i++){
                for(j=0; j<m ;j++)
                        printf("%d ",mat[i][j]);
                printf("\n");
        }
}

void create_matrices(int** &mat_a,int** &d_mat_a,int** &dd_mat_a,int n,int m)
{
        int i;

        int size_row = sizeof(int*) * n;
        int size_col = sizeof(int ) * m;

        create_host_matrix(&mat_a,n,m);
        fill_host_matrix(mat_a,n,m);

        d_mat_a = (int**) malloc(size_row);
        hipMalloc((void**)& d_mat_a[0], sizeof(int) * m * n );
        hipMemcpy(d_mat_a[0], mat_a[0], sizeof(int) * m * n ,hipMemcpyHostToDevice);

        for(i=1;i<n;i++){
                d_mat_a[i]=(d_mat_a[i-1]+m);
        }

        hipMalloc((void***)&dd_mat_a,size_row);
        hipMemcpy(dd_mat_a,d_mat_a,size_row,hipMemcpyHostToDevice);

}

int main()
{

        int** mat_a; int** d_mat_a;      int** dd_mat_a;
        int** mat_b; int** d_mat_b;      int** dd_mat_b;
        int** mat_c; int** d_mat_c;      int** dd_mat_c;

        int i;
        int size_row = sizeof(int*) * N;
        int size_col = sizeof(int ) * N;


        create_matrices(mat_a,d_mat_a,dd_mat_a,N,N);
        create_matrices(mat_b,d_mat_b,dd_mat_b,N,N);
        create_matrices(mat_c,d_mat_c,dd_mat_c,N,N);

        printf("Matrix A\n");
        print(mat_a,N,N);
        printf("\n");

        printf("Matrix B\n");
        
        print(mat_b,N,N);
        printf("\n");

        dim3 my_block(THREADS_PER_BLOCK,THREADS_PER_BLOCK);
        dim3 my_grid((N + my_block.x-1)/my_block.x, (N + my_block.y-1)/my_block.y);


        //matrix_mult<<<my_grid,my_block>>>(dd_mat_a,dd_mat_b,dd_mat_c,N);
        matrix_mult_shared<<<my_grid,my_block>>>(dd_mat_a,dd_mat_b,dd_mat_c,N);
        for(i=0;i<N;i++){
                hipMemcpy(mat_c[i],d_mat_c[i],size_col,hipMemcpyDeviceToHost);
        }

        printf("\n");
        printf("Matrix C \n");
        print(mat_c,N,N);
        return 0;
}
