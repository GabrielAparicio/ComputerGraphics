
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

#define N 20
#define M 3


__global__ void sum(int *a, int *b, int *c, int n) {

	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if (index < n)
 		c[index] = a[index] + b[index];
}

void fill_matrix(int *a, int n){
	int i;
	for(i=0;i<n;i++){
		a[i]=rand()%99;
	}
}

void print_matrix(int *a, int n){
	int i;
	for(i=0;i<n;i++){
		printf("%d ",a[i]);
	}
	printf("\n");
}

int main() 	
{

	int *a, *b, *c;
	int *d_a, *d_b, *d_c; 
	int size = N * sizeof(int);

	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);

	a = (int *)malloc(size); 
	fill_matrix(a, N);

	b = (int *)malloc(size);
 	fill_matrix(b, N);

	c = (int *)malloc(size);
	print_matrix(a,N);
	print_matrix(b,N); 

	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

	sum<<<(N + M-1) / M,M>>>(d_a, d_b, d_c, N);

	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

	print_matrix(c,N);

	free(a); 
	free(b); 
	free(c);

	hipFree(d_a); 
	hipFree(d_b); 
	hipFree(d_c);

	return 0;
	
}

